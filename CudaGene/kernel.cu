#include "hip/hip_runtime.h"
﻿#pragma once
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] - b[i];
}


extern "C" void
launch_addkernel(int size, int *c, const int *a, const int *b) {
    addKernel << <1, size >> > (c, a, b);
}

